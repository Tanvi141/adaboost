#include "hip/hip_runtime.h"
#ifndef TEST_CUDA_ADABOOST_CORE_OPERATIONS_IMPL_CU
#define TEST_CUDA_ADABOOST_CORE_OPERATIONS_IMPL_CU

#include<gtest/gtest.h>
#include<adaboost/cuda/core/cuda_data_structures.hpp>
#include<adaboost/cuda/utils/cuda_wrappers.hpp>
#include<adaboost/cuda/core/operations.cu>
#include<stdexcept>


__device__ float square_1_in(float x)
{
    return -x*x;
}

__device__  adaboost::cuda::core::func_t<float,float> p_func_here = square_1_in;

TEST(Cuda, Argmax)
{
    adaboost::utils::cuda::cuda_event_t has_happened;
    adaboost::utils::cuda::cuda_event_create(&has_happened);
    adaboost::cuda::core::VectorGPU<float> vec_i(5);
    vec_i.set(0, (float)-1.0);
    vec_i.set(1, (float)0.);
    vec_i.set(2, (float)1.);
    vec_i.set(3, (float)2.);
    vec_i.set(4, (float)3.);
    unsigned int block_size = 2;
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    unsigned result_gpu;

    adaboost::cuda::core::func_t <float, float> h_func;
    // adaboost::utils::cuda::cuda_malloc((void**)&h_func, sizeof(func_t <data_type_vec, data_type_ret>));
    
    // hipMemcpyToSymbol(HIP_SYMBOL(h_func), &p_func, sizeof(func_t <data_type_vec, data_type_ret>), 0, hipMemcpyHostToDevice);
    // hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(p_func_here), sizeof(adaboost::cuda::core::func_t <float, float>));
    // hipError_t err = hipGetLastError();        // Get error code
    // if ( err != hipSuccess ){
    //     printf("CUDA Error: %s\n", hipGetErrorString(err));
    //         exit(-1);
    // }
    adaboost::cuda::core::Argmax(square_1_in, vec_i, result_gpu, block_size);
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    EXPECT_EQ(4, result_gpu)<<"The arg max value is at 4.";
}
#endif