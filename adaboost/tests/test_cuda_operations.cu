#include "hip/hip_runtime.h"
#ifndef TEST_CUDA_ADABOOST_CORE_OPERATIONS_IMPL_CU
#define TEST_CUDA_ADABOOST_CORE_OPERATIONS_IMPL_CU

#include<gtest/gtest.h>
#include<adaboost/cuda/core/cuda_data_structures.hpp>
#include<adaboost/cuda/utils/cuda_wrappers.hpp>
#include<adaboost/cuda/core/operations.cu>
#include<stdexcept>


__device__ float square_1_in(float x)
{
    return -x*x;
}

__device__  adaboost::cuda::core::func_t<float,float> p_func_here = square_1_in;

TEST(Cuda, Argmax)
{
    adaboost::utils::cuda::cuda_event_t has_happened;
    adaboost::utils::cuda::cuda_event_create(&has_happened);
    adaboost::cuda::core::VectorGPU<float> vec_i(10);
    vec_i.set(0, (float)-1.0);
    vec_i.set(1, (float)0.);
    vec_i.set(2, (float)1.);
    vec_i.set(3, (float)2.);
    vec_i.set(4, (float)3.);
    vec_i.set(5, (float)4.);
    vec_i.set(6, (float)9.);
    vec_i.set(7, (float)12.);
    vec_i.set(8, (float)8.);
    vec_i.set(9, (float)6.);
    unsigned int block_size = 3;
    unsigned int grid_size = 2;
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    unsigned result_gpu;
    vec_i.copy_to_device();
    adaboost::cuda::core::Argmax(square_1_in, vec_i, result_gpu, grid_size, block_size);
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    EXPECT_EQ(7, result_gpu)<<"The arg max value is at 7.";
}
#endif